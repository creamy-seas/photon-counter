#include "hip/hip_runtime.h"
/*
 * Data from the digitiser will be fed in as a block of R * (N * P=SP)
 *  - SP: samples
 *  - R: Number of repititions
 *  - N: Number of pulses
 *  - P: Period of a pulse
 *
 *  e.g.
 *  a1 a2 a3 a4 ... b1 b2 b3 b4 ... c1 c2 c3 c4 ...
 *
 *  and after evaluating at each point, will be mapped to a 2D array
 *
 *  a1 a2 a3 -> main_axis (sp_coordinate)
 *  b1 b2 b3 ...
 *  c1 c2 c3 ...
 *  d1 d2 d3 ...
 *  e1 e2 e3 ...
 *  f1 f2 f3 ...
 *  g1 g2 g3 ...
 *
 *  |
 *  rep-axis (r_coordinate)
 *
 *  And reduced to the following by summing up over the rep axis
 *  <1> <2> <3> ...
 */

#include <stdexcept>
#include <stdio.h>
#include <string>

#include "colours.hpp"
#include "power_kernel.hpp"

// Background signal copied once to GPU
__constant__ short gpu_chA_background[SP_POINTS];
__constant__ short gpu_chB_background[SP_POINTS];

/*
  Call function to copy background for each channel a single time:
  * - gpu_chA_background
  * - gpu_chB_background
  */
void GPU::copy_background_arrays_to_gpu(short *chA_background, short *chB_background) {

    int success = 0;
    success += hipMemcpyToSymbol(HIP_SYMBOL(gpu_chA_background), chA_background,
                                  SP_POINTS*sizeof(short));
    success += hipMemcpyToSymbol(HIP_SYMBOL(gpu_chB_background), chB_background,
                                  SP_POINTS*sizeof(short));
    if (success != 0)
        FAIL("Failed to copy background data TO the GPU - check that the arrays have SP_POINTS in them!");
}

__device__ void reduction_sum(
    unsigned long int* chA_cumulative_array,
    unsigned long int* chB_cumulative_array,
    unsigned long int* chAsq_cumulative_array,
    unsigned long int* chBsq_cumulative_array,
    int length
    ){
    /*
     * Reduce the array by summing up the total of each array into the first cell.
     */

    int idx = length / 2;
    int r_coordinate;

    while (idx != 0) {
        r_coordinate = threadIdx.x;
        while (r_coordinate < length){
            if (r_coordinate < idx) {
                chA_cumulative_array[r_coordinate] += chA_cumulative_array[r_coordinate + idx];
                chB_cumulative_array[r_coordinate] += chB_cumulative_array[r_coordinate + idx];
                chAsq_cumulative_array[r_coordinate] += chAsq_cumulative_array[r_coordinate + idx];
                chBsq_cumulative_array[r_coordinate] += chBsq_cumulative_array[r_coordinate + idx];
            }
            r_coordinate += blockDim.x;
        }
        __syncthreads();
        idx /= 2;
    }
}

///////////////////////////////////////////////////////////////////////////////
//          Power Kernel: Single copy of data to GPU + shared memory         //
///////////////////////////////////////////////////////////////////////////////
__global__ void power_kernel_runner(short *chA_data, short *chB_data,
                                    double *chA_out, double *chB_out,
                                    double *chAsq_out, double *chBsq_out){

    // An unsigned long int will be able to hold 2^32/(2^14) = 2^18 = 262144 data points from the 14bit digitiser
    // Since the SP digitizer will have a maximum of 254200 record (using the GetMaxNofRecordsFromNofSamples(adq_cu_ptr, 1))
    // This will be able to contain everything
    __shared__ unsigned long int chA_cumulative_array[R_POINTS];
    __shared__ unsigned long int chB_cumulative_array[R_POINTS];
    __shared__ unsigned long int chAsq_cumulative_array[R_POINTS];
    __shared__ unsigned long int chBsq_cumulative_array[R_POINTS];

    int sp_coordinate = blockIdx.x;
    int r_coordinate, coordinate;

    // Each block deals with a specific SP_POINT
    // Each thread iterates over R_POINTS for each SP_POINT
    while (sp_coordinate < SP_POINTS) {
        r_coordinate = threadIdx.x;

        while (r_coordinate < R_POINTS) {
            coordinate = r_coordinate * SP_POINTS + sp_coordinate;

            chA_cumulative_array[r_coordinate] = chA_data[coordinate] - gpu_chA_background[sp_coordinate];
            chB_cumulative_array[r_coordinate] = chB_data[coordinate] - gpu_chB_background[sp_coordinate];

            chAsq_cumulative_array[r_coordinate] = chA_cumulative_array[r_coordinate] * chA_cumulative_array[r_coordinate];
            chBsq_cumulative_array[r_coordinate] = chB_cumulative_array[r_coordinate] * chB_cumulative_array[r_coordinate];

            // Once thread has completed, shift the
            // row index by the number of allocated
            // threads and continue summation
            r_coordinate += blockDim.x;
        }

        // Ensure that all threads have completed execution
        __syncthreads();

        // Summation
        reduction_sum(chA_cumulative_array,
                      chB_cumulative_array,
                      chAsq_cumulative_array,
                      chBsq_cumulative_array,
                      R_POINTS);
        chA_out[sp_coordinate] = (double)chA_cumulative_array[0] / R_POINTS;
        chB_out[sp_coordinate] = (double)chB_cumulative_array[0] / R_POINTS;
        chAsq_out[sp_coordinate] = (double)chAsq_cumulative_array[0] / R_POINTS;
        chBsq_out[sp_coordinate] = (double)chBsq_cumulative_array[0] / R_POINTS;

        // Shift by number of allocated blocks along main-axis
        sp_coordinate += gridDim.x;
    }
}

void GPU::V1::power_kernel(
    short *chA_data,
    short *chB_data,
    double **data_out,
    short ***gpu_in, double ***gpu_out){
    // ==> Ensure that allocate_memory has been called
    // ==> Ensure that background arrays (set to 0 for no correction) have been copied over

    // Copy input data over to GPU.
    // Dereference the gpu_ch? (which is the address where the GPU memory location is kept)
    // in order to get the actual memory location
    int success = 0;
    success += hipMemcpy(*gpu_in[CHA], chA_data,
                          R_POINTS * SP_POINTS * sizeof(short),
                          hipMemcpyHostToDevice);
    success += hipMemcpy(*gpu_in[CHB], chB_data,
                          R_POINTS * SP_POINTS * sizeof(short),
                          hipMemcpyHostToDevice);
    if (success != 0) FAIL("Failed to copy data TO the GPU!");

    // Run kernel
    power_kernel_runner<<<BLOCKS, THREADS_PER_BLOCK>>>(*gpu_in[CHA], *gpu_in[CHB],
                                                       *gpu_out[CHA], *gpu_out[CHB],
                                                       *gpu_out[CHASQ], *gpu_out[CHBSQ]);
    // Copy from device
    success += hipMemcpy(data_out[CHA], *gpu_out[CHA],
                          SP_POINTS * sizeof(double),
                          hipMemcpyDeviceToHost);
    success += hipMemcpy(data_out[CHB], *gpu_out[CHB],
                          SP_POINTS * sizeof(double),
                          hipMemcpyDeviceToHost);
    success += hipMemcpy(data_out[CHASQ],*gpu_out[CHASQ],
                          SP_POINTS * sizeof(double),
                          hipMemcpyDeviceToHost);
    success += hipMemcpy(data_out[CHBSQ], *gpu_out[CHBSQ],
                          SP_POINTS * sizeof(double),
                          hipMemcpyDeviceToHost);
    if (success != 0) FAIL("Failed to copy data FROM the GPU!");

    // Manually evaluate sq = chAsq + chBsq
    for (int i(0); i < SP_POINTS; i++)
        data_out[SQ][i] = data_out[CHASQ][i] + data_out[CHBSQ][i];

    // Ensure that free_memory is called ==>
}

__global__ void power_kernel_runner_v2(short *chA_data, short *chB_data,
                                       double *chA_out, double *chB_out,
                                       double *chAsq_out, double *chBsq_out){

    // An unsigned long int will be able to hold 2^32/(2^14) = 2^18 = 262144 data points from the 14bit digitiser
    // Since the SP digitizer will have a maximum of 254200 record (using the GetMaxNofRecordsFromNofSamples(adq_cu_ptr, 1))
    // This will be able to contain everything
    __shared__ unsigned long int chA_cumulative_array[R_POINTS_PER_CHUNK];
    __shared__ unsigned long int chB_cumulative_array[R_POINTS_PER_CHUNK];
    __shared__ unsigned long int chAsq_cumulative_array[R_POINTS_PER_CHUNK];
    __shared__ unsigned long int chBsq_cumulative_array[R_POINTS_PER_CHUNK];

    int sp_coordinate = blockIdx.x;
    int r_coordinate, coordinate;

    // Each block deals with a specific SP_POINT
    // Each thread iterates over R_POINTS_PER_CHUNK for each SP_POINT
    while (sp_coordinate < SP_POINTS) {
        r_coordinate = threadIdx.x;

        while (r_coordinate < R_POINTS_PER_CHUNK) {
            coordinate = r_coordinate * SP_POINTS + sp_coordinate;

            chA_cumulative_array[r_coordinate] = chA_data[coordinate] - gpu_chA_background[sp_coordinate];
            chB_cumulative_array[r_coordinate] = chB_data[coordinate] - gpu_chB_background[sp_coordinate];

            chAsq_cumulative_array[r_coordinate] = chA_cumulative_array[r_coordinate] * chA_cumulative_array[r_coordinate];
            chBsq_cumulative_array[r_coordinate] = chB_cumulative_array[r_coordinate] * chB_cumulative_array[r_coordinate];

            // Once thread has completed, shift the
            // row index by the number of allocated
            // threads and continue summation
            r_coordinate += blockDim.x;
        }

        // Ensure that all threads have completed execution
        __syncthreads();

        // Summation
        reduction_sum(chA_cumulative_array,
                      chB_cumulative_array,
                      chAsq_cumulative_array,
                      chBsq_cumulative_array,
                      R_POINTS_PER_CHUNK);
        chA_out[sp_coordinate] = (double)chA_cumulative_array[0] / R_POINTS_PER_CHUNK;
        chB_out[sp_coordinate] = (double)chB_cumulative_array[0] / R_POINTS_PER_CHUNK;
        chAsq_out[sp_coordinate] = (double)chAsq_cumulative_array[0] / R_POINTS_PER_CHUNK;
        chBsq_out[sp_coordinate] = (double)chBsq_cumulative_array[0] / R_POINTS_PER_CHUNK;

        // Shift by number of allocated blocks along main-axis
        sp_coordinate += gridDim.x;
    }
}

void GPU::V2::power_kernel(
    short *chA_data,
    short *chB_data,
    double **data_out,
    short ***gpu_in0, short ***gpu_in1,
    double ***gpu_out0, double ***gpu_out1,
    double ***cpu_out
    ){
    // ==> Ensure that allocate_memory has been called
    // ==> Ensure that background arrays (set to 0 for no correction) have been copied over

    // Launch two streams, dealing with alternating chunks
    // steam0       stream1     stream0      stream1
    // a1a2a3a4.... b1b2b3b4... c1c2c3c4.... d1d2d3d4...
    //
    // - Each chunk has length SP_POINTS
    // - There are R_POINTS/R_POINTS_PER_CHUNK total chunks to iterate through

    // Note that we need to dereference the addresses of the GPU memory pointer e.g. *gpu_in0[CJA]
    // In order to get the memory location on the gpu

    int odx; // ouput index
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0); hipStreamCreate(&stream1);
    for (int  rep(0); rep < R_POINTS; rep+=2*R_POINTS_PER_CHUNK){
        // Memcpy -> Kernel -> Memcpy must be in the order below (breadth first)
        // to prevent blocking of kernel execution by batching of similar commands

        // Copy input data over to GPU asynchornously in chunks
        // Dereference the gpu_ch? (which is the address where the GPU memory location is kept)
        // in order to get the actual memory location
        hipMemcpyAsync(
            *gpu_in0[CHA], chA_data + rep*SP_POINTS,
            R_POINTS_PER_CHUNK * SP_POINTS * sizeof(short),
            hipMemcpyHostToDevice,
            stream0);
        hipMemcpyAsync(
            *gpu_in1[CHA], chA_data + (rep+R_POINTS_PER_CHUNK)*SP_POINTS,
            R_POINTS_PER_CHUNK * SP_POINTS * sizeof(short),
            hipMemcpyHostToDevice,
            stream1);
        hipMemcpyAsync(
            *gpu_in0[CHB], chB_data + rep*SP_POINTS,
            R_POINTS_PER_CHUNK * SP_POINTS * sizeof(short),
            hipMemcpyHostToDevice,
            stream0);
        hipMemcpyAsync(
            *gpu_in1[CHB], chB_data + (rep+R_POINTS_PER_CHUNK)*SP_POINTS,
            R_POINTS_PER_CHUNK * SP_POINTS * sizeof(short),
            hipMemcpyHostToDevice,
            stream1);

        // Run kernel(s)
        power_kernel_runner_v2<<<BLOCKS, THREADS_PER_BLOCK, 0, stream0>>>(
            *gpu_in0[CHA], *gpu_in0[CHB],
            *gpu_out0[CHA], *gpu_out0[CHB],
            *gpu_out0[CHASQ], *gpu_out0[CHBSQ]);

        power_kernel_runner_v2<<<BLOCKS, THREADS_PER_BLOCK, 0, stream1>>>(
            *gpu_in1[CHA], *gpu_in1[CHB],
            *gpu_out1[CHA], *gpu_out1[CHB],
            *gpu_out1[CHASQ], *gpu_out1[CHBSQ]);

        // Copy from GPU to CPU
        for (int k(0); k < GPU::no_outputs_from_gpu; k++){
            odx = GPU::outputs_from_gpu[k];
            hipMemcpyAsync(
                *cpu_out[odx] + rep*SP_POINTS,
                *gpu_out0[odx],
                SP_POINTS * sizeof(double),
                hipMemcpyDeviceToHost,
                stream0);
            hipMemcpyAsync(
                *cpu_out[odx] + (rep+1)*SP_POINTS,
                *gpu_out1[odx],
                SP_POINTS * sizeof(double),
                hipMemcpyDeviceToHost,
                stream1);
        }
    }
    // Ensure that execution of each stream finishes
    hipStreamSynchronize(stream0); hipStreamSynchronize(stream1);
    hipStreamDestroy(stream0); hipStreamDestroy(stream1);

    // Sum on CPU, collecting up the separate chunks into final output
    // data_out[CHB][0] = (*cpu_out[CHB])[2];
    for (int sp(0); sp < SP_POINTS; sp++){
        for (int k(0); k < GPU::no_outputs_from_gpu; k++){
            odx = GPU::outputs_from_gpu[k];
            data_out[k][sp] = 0;
            for (int rep(0); rep < R_POINTS; rep+=2*R_POINTS_PER_CHUNK){
                data_out[odx][sp] += ((*cpu_out)[odx][sp+rep*SP_POINTS] + (*cpu_out)[odx][sp+(rep+1)*SP_POINTS]) / 2;
            }
        }
        data_out[SQ][sp] = data_out[CHBSQ][sp];// + data_out[CHBSQ][sp];
    }

    // Ensure that free_memory is called ==>
}
