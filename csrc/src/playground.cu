#include "hip/hip_runtime.h"
#include <iostream>
#include "playground.hpp"
#include "colours.hpp"

__global__ void example_gpu_func_kernel(int a, int b, float *c){
        *c = (float)(a * a + b * b);
}

float GPU::example_gpu_func(short a, short b) {

        // hipDeviceProp_t prop = fetch_gpu_parameters();

        /* hipDeviceProp_t program_prop; */
        /* memset(&program_prop, 0, sizeof(hipDeviceProp_t)); */
        /* program_prop.maxGridSize[0] = 100; */

        // Allocate device and host variables
        float c;
        float *dev_c;

        // Memory allocation on device
        hipMalloc((void**) &dev_c, sizeof(float));

        // Kernel invoction
        example_gpu_func_kernel<<<1,1>>>(a, b, dev_c);

        // Copy back to device
        hipMemcpy(
                &c,
                dev_c,
                sizeof(float),
                hipMemcpyDeviceToHost
                );


        hipFree(dev_c);
        OKGREEN("GPU KERNEL Complete!");

        std::cout << c << std::endl;

        return c;
}
