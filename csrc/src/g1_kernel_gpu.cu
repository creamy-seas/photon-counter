#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sstream>

#include <iostream>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
// #include <helper_functions.h>

#include "g1_kernel.hpp"
#include "logging.hpp"

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE 50
#define FILTER_KERNEL_SIZE 11

// Complex data type
typedef float2 Complex;

static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}
static __global__ void ComplexPointwiseMulAndScale(
    Complex *a, const Complex *b,
    int size, float scale) {

    const int step = blockDim.x * gridDim.x;
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = thread_id; i < size; i += step)
        a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
}

// static __device__ __host__ inline hipfftComplex complex_multiplication(hipfftComplex a, hipfftComplex b) {
//     Complex c;
//     c.x = a.x * b.x - a.y * b.y;
//     c.y = a.x * b.y + a.y * b.x;
//     return c;
// }
static __global__ void fftw_square(hipfftComplex *fourier_transform, float normalisation) {

    const int step = blockDim.x * gridDim.x;
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    int mid_point = (G1_DIGITISER_POINTS / 2) + 1;
    for (int i = thread_id; i < mid_point; i += step) {
        // fourier_transform[i] = complex_multiplication(fourier_transform[i], fourier_transform[i]);
        fourier_transform[i].x = (
            fourier_transform[i].x * fourier_transform[i].x +
            fourier_transform[i].y * fourier_transform[i].y) / normalisation;
        fourier_transform[i].y = 0;
    }
    // fourier_transform[0].x = 0;
}

// Computes convolution on the host
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal) {
    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;

    // Loop over output element indices
    for (int i = 0; i < signal_size; ++i) {
        filtered_signal[i].x = filtered_signal[i].y = 0;

        // Loop over convolution indices
        for (int j = -maxRadius + 1; j <= minRadius; ++j) {
            int k = i + j;

            if (k >= 0 && k < signal_size) {
                filtered_signal[i] =
                    ComplexAdd(filtered_signal[i],
                               ComplexMul(signal[k], filter_kernel[minRadius - j]));
            }
        }
    }
}

int PadData(
    const Complex *signal, Complex **padded_signal, int signal_size,
    const Complex *filter_kernel, Complex **padded_filter_kernel,
    int filter_kernel_size) {

    int minRadius = filter_kernel_size / 2;
    int maxRadius = filter_kernel_size - minRadius;
    int new_size = signal_size + maxRadius;

    // Pad signal
    Complex *new_data =
        reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
    memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
    memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
    *padded_signal = new_data;

    // Pad filter
    new_data = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * new_size));
    memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
    memset(new_data + maxRadius, 0,
           (new_size - filter_kernel_size) * sizeof(Complex));
    memcpy(new_data + new_size - minRadius, filter_kernel,
           minRadius * sizeof(Complex));
    *padded_filter_kernel = new_data;

    return new_size;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest() {
    printf("[simpleCUFFT] is starting...\n");

    // Allocate host memory for the signal
    Complex *h_signal = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));
    for (unsigned int i = 0; i < SIGNAL_SIZE; ++i){
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX); h_signal[i].y = 0;
    }


    // Allocate host memory for the filter
    Complex *h_filter_kernel = reinterpret_cast<Complex *>(malloc(sizeof(Complex) * FILTER_KERNEL_SIZE));
    for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i){
        h_filter_kernel[i].x = rand() / static_cast<float>(RAND_MAX); h_filter_kernel[i].y = 0;
    }

    // Pad signal and filter kernel
    Complex *h_padded_signal;
    Complex *h_padded_filter_kernel;
    int new_size =
        PadData(h_signal, &h_padded_signal, SIGNAL_SIZE, h_filter_kernel,
                &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
    int mem_size = sizeof(Complex) * new_size;

    // Allocate device memory for signal
    Complex *d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_signal), mem_size));
    checkCudaErrors(
        hipMemcpy(d_signal, h_padded_signal, mem_size, hipMemcpyHostToDevice));

    // Allocate device memory for filter kernel
    Complex *d_filter_kernel;
    checkCudaErrors(
        hipMalloc(reinterpret_cast<void **>(&d_filter_kernel), mem_size));
    checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
                               hipMemcpyHostToDevice));

    // CUFFT plan simple API
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, new_size, HIPFFT_C2C, 1));

    // Transform signal and kernel
    printf("Transforming signal hipfftExecC2C\n");
    checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                                 reinterpret_cast<hipfftComplex *>(d_signal),
                                 HIPFFT_FORWARD));

    // Multiply the coefficients together and normalize the result
    printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
    ComplexPointwiseMulAndScale<<<32, 256>>>(d_signal, d_filter_kernel, new_size,
                                             1.0f / new_size);

    // Transform signal back
    printf("Transforming signal back hipfftExecC2C\n");
    checkCudaErrors(hipfftExecC2C(plan, reinterpret_cast<hipfftComplex *>(d_signal),
                                 reinterpret_cast<hipfftComplex *>(d_signal),
                                 HIPFFT_BACKWARD));

    // Copy device memory to host
    Complex *h_convolved_signal = h_padded_signal;
    checkCudaErrors(hipMemcpy(h_convolved_signal, d_signal, mem_size,
                               hipMemcpyDeviceToHost));

    // Convolve on the host
    Complex *h_convolved_signal_ref =
        reinterpret_cast<Complex *>(malloc(sizeof(Complex) * SIGNAL_SIZE));
    Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel, FILTER_KERNEL_SIZE,
             h_convolved_signal_ref);

    // Destroy CUFFT context
    checkCudaErrors(hipfftDestroy(plan));

    free(h_signal);
    free(h_filter_kernel);
    free(h_padded_signal);
    free(h_padded_filter_kernel);
    free(h_convolved_signal_ref);
    checkCudaErrors(hipFree(d_signal));
    checkCudaErrors(hipFree(d_filter_kernel));

    OKGREEN("Done");

    // hipfftExecR2C
    // Square and pivot
    // hipfftExecC2R
}

int G1::GPU::g1_prepare_fftw_plan(hipfftHandle *&plans_forward, hipfftHandle *&plans_backward) {

    OKBLUE("Generating optimised forward and backward plans");
    plans_forward = new hipfftHandle[G1::no_outputs]; plans_backward = new hipfftHandle[G1::no_outputs];
    for (int i(0); i < G1::no_outputs; i++) {
        if (
            hipfftPlan1d(&plans_forward[i], G1_DIGITISER_POINTS, HIPFFT_R2C, 1) != HIPFFT_SUCCESS)
            FAIL("Failed to create FFTW Forward Plan on GPU");
        if (
            hipfftPlan1d(&plans_backward[i], G1_DIGITISER_POINTS, HIPFFT_C2R, 1) != HIPFFT_SUCCESS)
            FAIL("Failed to create FFTW Backward Plan on GPU");
    }
    return 0;
}

void G1::GPU::allocate_memory(short *&chA_data, short *&chB_data,
                              hipfftReal **&gpu_inout, hipfftComplex **&gpu_aux,
                              float **&cpu_out) {
    int success = 0;
    success += hipHostAlloc(reinterpret_cast<void**>(&chA_data),
                             SP_POINTS * R_POINTS * sizeof(short),
                             hipHostMallocDefault);
    success += hipHostAlloc(reinterpret_cast<void**>(&chB_data),
                             SP_POINTS * R_POINTS * sizeof(short),
                             hipHostMallocDefault);
    if (success != 0) FAIL("G1 Kernel: Failed to allocate locked input memory on CPU.");

    gpu_inout = new hipfftReal*[G1::no_outputs];
    gpu_aux = new hipfftComplex*[G1::no_outputs];
    for (int i(0); i < G1::no_outputs; i++){
        success += hipMalloc(reinterpret_cast<void**>(&gpu_inout[i]), G1_DIGITISER_POINTS * sizeof(hipfftReal));
        success += hipMalloc(reinterpret_cast<void**>(&gpu_aux[i]), (int(G1_DIGITISER_POINTS / 2) + 1) * sizeof(hipfftComplex));
    }
    if (success != 0) FAIL("G1 Kernel: Failed to allocate memory on GPU.");

    cpu_out = new float*[G1::no_outputs];
    for (int i(0); i < G1::no_outputs; i++)
        success += hipHostAlloc(reinterpret_cast<void**>(&cpu_out[i]), G1_DIGITISER_POINTS * sizeof(float), hipHostMallocDefault);
    if (success != 0) FAIL("G1 Kernel: Failed to allocate locked output memory on CPU.");
}

void G1::GPU::free_memory(short *chA_data, short *chB_data,
                          hipfftReal **gpu_inout, hipfftComplex **gpu_aux,
                          float **cpu_out) {
    OKBLUE("G1 Kernel: Deallocating memory on GPU and CPU.");
    int success = 0;
    success += hipHostFree(chA_data);
    success += hipHostFree(chB_data);
    if (success != 0) FAIL("Power Kernel: Failed to free locked input memory on CPU.");

    for (int i(0); i < G1::no_outputs; i++) {
        success += hipFree(gpu_inout[i]);
        success += hipFree(gpu_aux[i]);
    }
    delete[] gpu_inout;
    delete[] gpu_aux;
    if (success != 0) FAIL("Power Kernel: Failed to free memory on GPU.");

    for (int i(0); i < G1::no_outputs; i++) {
        success += hipHostFree(cpu_out[i]);
    }
    delete[] cpu_out;
    if (success != 0) FAIL("Power Kernel: Failed to free output memory on CPU.");
}

void handle_error(hipfftResult result, std::string error_message){
    if (result != HIPFFT_SUCCESS) {
        std::stringstream ss;
        ss << error_message << ": Error code " << result << "\nCheck https://docs.nvidia.com/cuda/cufft/index.html#cufftresult";
        FAIL(ss.str());
    }
}
void handle_error(hipError_t result, std::string error_message){
    std::cout << result << std::endl;

    if (result != 0) {
        std::stringstream ss;
        ss << error_message << ": Error code " << result << "\nCheck  https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html#group__CUDART__TYPES_1g3f51e3575c2178246db0a94a430e0038";
        FAIL(ss.str());
    }
}

void G1::GPU::g1_kernel(
    float **preprocessed_data, float *variance_list,
    hipfftReal **gpu_inout, hipfftComplex **gpu_aux, float **cpu_out,
    hipfftHandle *plans_forward, hipfftHandle *plans_backward){

    // Copy data
     checkCudaErrors(hipMemcpy(gpu_inout[CHAG1], preprocessed_data[CHAG1],
                                sizeof(float) * G1_DIGITISER_POINTS, hipMemcpyHostToDevice));

     // Forward transform
     checkCudaErrors(hipfftExecR2C(
                         plans_forward[0],
                         gpu_inout[CHAG1], gpu_aux[CHAG1]));
     // Square and normalise
     fftw_square<<<G1_DIGITISER_POINTS / 1024 + 1,1024>>>(
         gpu_aux[CHAG1],
         G1_DIGITISER_POINTS * G1_DIGITISER_POINTS * variance_list[CHAG1]);
     // Backward transform
     checkCudaErrors(hipfftExecC2R(
                         plans_backward[0],
                         gpu_aux[CHAG1], gpu_inout[CHAG1]));

     // Copy back to CPU
     checkCudaErrors(
         hipMemcpy(cpu_out[CHAG1], gpu_inout[CHAG1],
                    sizeof(hipfftReal) * G1_DIGITISER_POINTS,
                    hipMemcpyDeviceToHost));
}
